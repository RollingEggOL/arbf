#include "hip/hip_runtime.h"
//
//  kernel.cu
//  arbf-image-cuda
//
//  Created by Ke Liu on 10/01/14.
//  Copyright (c) 2014 Ke Liu. All rights reserved.
//

#include <vector>
#include <set>
#include <hip/hip_runtime.h>
#include "../include/common.h"

extern std::vector<PixelTag> g_pixel_tag;
extern SurFaceMesh* samples;
extern DBL7VECT* centers;
extern int X;
extern int Y;
extern double *g_F;

extern std::vector<std::vector<int> > vertex_faces_1ring;
extern std::vector<std::set<int> > neigh_vertex_1ring;
extern std::vector<std::set<int> > face_faces_1ring;
#ifdef NEIGHBOR_2RING
extern std::vector<std::vector<int> > vertex_faces_2ring;
extern std::vector<std::set<int> > neigh_vertex_2ring;
extern std::vector<std::set<int> > face_faces_2ring;
#endif

#define mat_elem(a, y, x, n) (a + ((y) * (n) + (x)))

class vector2d
{
public:
	__device__ vector2d(double d1, double d2)
	{
		_data.x = make_double2(d1, d2).x;
		_data.y = make_double2(d1, d2).y;
	}

	// getter
	__device__ const double2 data() const
	{
		return _data;
	}

	__device__ double x() const
	{
		return _data.x;
	}

	__device__ double y() const
	{
		return _data.y;
	}

	// scalar multiplication
	__device__ vector2d& mul(double s)
	{
		_data.x *= s;
		_data.y *= s;

		return *this;
	}

	// norm of vector of size 2
	__device__ double norm()
	{
		return sqrt(SQ(_data.x) + SQ(_data.y));
	}

	// dot product of vector of size 2
	__device__ double dot(const vector2d& v2)
	{
		return _data.x * v2.x() + _data.y * v2.y();
	}

private:
	double2 _data;
};

// swap row r1 and r2 of matrix A and corresponding elements of vector b
__device__ void swap_row(double *A, double *b, int r1, int r2, int n)
{
	double tmp, *p1, *p2;
	int i;
	
	if (r1 == r2)
		return;

	for (i = 0; i < n; i++)
	{
		p1 = mat_elem(A, r1, i, n);
		p2 = mat_elem(A, r2, i, n);
		tmp = *p1;
		*p1 = *p2;
		*p2 = tmp;
	}
	tmp = b[r1];
	b[r1] = b[r2];
	b[r2] = tmp;
}

// solve linear system A*x = b
__device__ void solve(double *x, double* A, double* b, int n)
{
#define A(y, x) (*mat_elem(A, y, x, n))
	int maxRow, row, col, dia, j;
	double max, tmp;

	// Gaussian elimination
	for (dia = 0; dia < n; dia++)
	{
		// find maxRow and max diagonal element
		maxRow = dia;
		max = A(dia, dia);
		for (row = dia + 1; row < n; row++)
		{
			if ((tmp = fabs(A(row, dia))) > max)
			{
				maxRow = row;
				max = tmp;
			}
		}
		swap_row(A, b, dia, maxRow, n);

		// eleminate
		for (row = dia + 1; row < n; row++)
		{
			tmp = A(row, dia) / A(dia, dia);
			for (col = dia + 1; col < n; col++)
				A(row, col) -= (tmp * A(dia, col));
			A(row, dia) = 0.0;
			b[row] -= (tmp * b[dia]);
		}
	}

	// back-substitution
	for (row = n - 1; row >= 0; row--)
	{
		tmp = b[row];
		for (j = n - 1; j > row; j--)
			tmp -= (x[j] * A(row, j));
		x[row] = tmp / A(row, row);
	}
#undef A
}

__device__ bool is_in_triangle(int _xx, int _xy, double _ax, double _ay, double _bx, double _by, double _cx, double _cy)
{
	vector2d ab(_bx - _ax, _by - _ay);
	vector2d bc(_cx - _bx, _cy - _by);
	vector2d ca(_ax - _cx, _ay - _cy);
	vector2d ax(_xx - _ax, _xy - _ay);
	vector2d bx(_xx - _bx, _xy - _by);
	vector2d cx(_xx - _cx, _xy - _cy);

	double area = 0.0, area1 = 0.0, area2 = 0.0, area3 = 0.0;
	double cosTheta = ab.dot(ca.mul(-1)) / (ab.norm()*ca.norm());
	area = 0.5*ab.norm()*ca.norm()*sqrt(1.0 - SQ(cosTheta));
	double cosTheta1 = ab.dot(ax) / (ab.norm()*ax.norm());
	area1 = 0.5*ab.norm()*ax.norm()*sqrt(1.0 - SQ(cosTheta1));
	double cosTheta2 = cx.dot(ca) / (cx.norm()*ca.norm());
	area2 = 0.5*cx.norm()*ca.norm()*sqrt(1.0 - SQ(cosTheta2));
	double cosTheta3 = bx.dot(bc) / (bx.norm()*bc.norm());
	area3 = 0.5*bx.norm()*bc.norm()*sqrt(1.0 - SQ(cosTheta3));

	if (isnan(area)) area = 0.0;
	if (isnan(area1)) area1 = 0.0;
	if (isnan(area2)) area2 = 0.0;
	if (isnan(area3)) area3 = 0.0;

	if (abs(area1 + area2 + area3 - area) < EPSILON)
		return true;
	else
		return false;
}

// distance under metric T
__device__ double compute_distance(const double *v0, const double *v1, const double *T)
{
	vector2d dis(v0[0] - v1[0], v0[1] - v1[1]);
	vector2d tmp(dis.x() * T[0] + dis.y() * T[2], dis.x() * T[1] + dis.y() * T[3]);
	
	return sqrt(dis.dot(tmp));
}

// basis function MQ
__device__ double phi(double r)
{
	return sqrt(SQ(r) + SQ(0.5));
}

// basis function IMQ
//__device__ double phi(double r)
//{
//    return 1.0 / (sqrt(SQ(r) + SQ(1.8)));
//}

// basis function Gaussian
//__device__ double phi(double r)
//{
//    double c = 0.01;
//    return exp(-SQ(c*r));
//}

// basis function TPS
//__device__ double phi(double r)
//{
//    if (abs(r) < EPSILON)
//        return 0;
//    else
//        return SQ(r) * log(r);
//}

__global__ void find_triangle_kernel(int* dx, int* dy, size_t stride, size_t blockStride,
	const DBL7VECT* vertex, const INT3VECT* face, int N)
{
	extern __shared__ int sdata[];

	int tx = blockIdx.x*blockDim.x + threadIdx.x;

	if (tx < N)
	{
		// face[tx].a = sdata[threadIdx.x*blockStride + 10]
		// face[tx].b = sdata[threadIdx.x*blockStride + 11]
		// face[tx].c = sdata[threadIdx.x*blockStride + 12]
		sdata[threadIdx.x*blockStride + 10] = face[tx].a;
		sdata[threadIdx.x*blockStride + 11] = face[tx].b;
		sdata[threadIdx.x*blockStride + 12] = face[tx].c;

		// v0x = sdata[threadIdx.x*blockStride]
		// v0y = sdata[threadIdx.x*blockStride+1]
		// v1x = sdata[threadIdx.x*blockStride+2]
		// v1y = sdata[threadIdx.x*blockStride+3]
		// v2x = sdata[threadIdx.x*blockStride+4]
		// v2y = sdata[threadIdx.x*blockStride+5]
		sdata[threadIdx.x*blockStride] = vertex[sdata[threadIdx.x*blockStride + 10]].x;
		sdata[threadIdx.x*blockStride + 1] = vertex[sdata[threadIdx.x*blockStride + 10]].y;
		sdata[threadIdx.x*blockStride + 2] = vertex[sdata[threadIdx.x*blockStride + 11]].x;
		sdata[threadIdx.x*blockStride + 3] = vertex[sdata[threadIdx.x*blockStride + 11]].y;
		sdata[threadIdx.x*blockStride + 4] = vertex[sdata[threadIdx.x*blockStride + 12]].x;
		sdata[threadIdx.x*blockStride + 5] = vertex[sdata[threadIdx.x*blockStride + 12]].y;

		// minX = sdata[threadIdx.x*blockStride+6]
		// maxX = sdata[threadIdx.x*blockStride+7]
		// minY = sdata[threadIdx.x*blockStride+8]
		// maxY = sdata[threadIdx.x*blockStride+9]
		sdata[threadIdx.x*blockStride + 6] = MIN(sdata[threadIdx.x*blockStride], sdata[threadIdx.x*blockStride + 2]);
		sdata[threadIdx.x*blockStride + 6] = MIN(sdata[threadIdx.x*blockStride + 6], sdata[threadIdx.x*blockStride + 4]);
		sdata[threadIdx.x*blockStride + 7] = MAX(sdata[threadIdx.x*blockStride], sdata[threadIdx.x*blockStride + 2]);
		sdata[threadIdx.x*blockStride + 7] = MAX(sdata[threadIdx.x*blockStride + 7], sdata[threadIdx.x*blockStride + 4]);
		sdata[threadIdx.x*blockStride + 8] = MIN(sdata[threadIdx.x*blockStride + 1], sdata[threadIdx.x*blockStride + 3]);
		sdata[threadIdx.x*blockStride + 8] = MIN(sdata[threadIdx.x*blockStride + 8], sdata[threadIdx.x*blockStride + 5]);
		sdata[threadIdx.x*blockStride + 9] = MAX(sdata[threadIdx.x*blockStride + 1], sdata[threadIdx.x*blockStride + 3]);
		sdata[threadIdx.x*blockStride + 9] = MAX(sdata[threadIdx.x*blockStride + 9], sdata[threadIdx.x*blockStride + 5]);

		int n = 0; // actual number of pixels in current triangle
		int k = 1; // indice of pixel coord

		for (int j = sdata[threadIdx.x*blockStride + 8]; j <= sdata[threadIdx.x*blockStride + 9]; ++j)
		{
			for (int i = sdata[threadIdx.x*blockStride + 6]; i <= sdata[threadIdx.x*blockStride + 7]; ++i)
			{
				int p1 = i;
				int p2 = j;
				if (is_in_triangle(p1, p2,
					sdata[threadIdx.x*blockStride], sdata[threadIdx.x*blockStride + 1], sdata[threadIdx.x*blockStride + 2], sdata[threadIdx.x*blockStride + 3],
					sdata[threadIdx.x*blockStride + 4], sdata[threadIdx.x*blockStride + 5]))
				{
					n++;
					dx[tx*stride] = n;
					dy[tx*stride] = n;
					dx[tx*stride + k] = i;
					dy[tx*stride + k] = j;
					k++;
				}
			}
		}
	}
}

__global__ void interpolate_kernel(double *F, int numFaces, int X, int Y, int numMetricElem, int maxNumNeighFace,
	const DBL7VECT* vertices, const INT3VECT* faces, const DBL7VECT* centers, const double *T, const int *face_face)
{
	extern __shared__ int s_face_face[]; // face_face on shared memory
	int tx = blockIdx.x * blockDim.x + threadIdx.x; // global thread ID
	int tid = threadIdx.x; // local thread ID
	int x[NUM_PIXEL_PER_FACE], y[NUM_PIXEL_PER_FACE]; // coords of pixels in current triangle

	if (tx < numFaces)
	{
		// find TRIANGLE <-> PIXEL mapping
		const INT3VECT *faceVertices = &(faces[tx]);
		const DBL7VECT *v0 = &(vertices[faceVertices->a]);
		const DBL7VECT *v1 = &(vertices[faceVertices->b]);
		const DBL7VECT *v2 = &(vertices[faceVertices->c]);
		double minX = MIN(MIN(v0->x, v1->x), v2->x);
		double maxX = MAX(MAX(v0->x, v1->x), v2->x);
		double minY = MIN(MIN(v0->y, v1->y), v2->y);
		double maxY = MAX(MAX(v0->y, v1->y), v2->y);
		int counter = 0; // stores how many pixels are actually in current face
		for (int j = (int) minY; j <= (int) ceil(maxY); ++j)
		{
			for (int i = (int) minX; i <= (int) ceil(maxX); ++i)
			{
				if (is_in_triangle(i, j, v0->x, v0->y, v1->x, v1->y, v2->x, v2->y))
				{
					x[counter] = i;
					y[counter] = j;
					counter ++;
				}
			}
		}

		// solve linear equations (ma * coeff = u)
		int startIdx_face = tid * maxNumNeighFace;
		s_face_face[startIdx_face] = tx * NUM_NEIGHBOR_TRIANGLES; // start index of current thread for face_face
		int numNeigh = face_face[s_face_face[startIdx_face]];
		double *ma = (double*) malloc(sizeof(double) * SQ(numNeigh)); // distance matrix, row-majored
		double *u = (double*) malloc(sizeof(double) * numNeigh); // right hand side vector
		double *coeff = (double*) malloc(sizeof(double) * numNeigh); // coefficient vector
		for (int i = 0; i < numNeigh; ++i)
		{
			s_face_face[startIdx_face + 1 + i] = face_face[s_face_face[startIdx_face] + 1 + i];
		}
		for (int i = 0; i < numNeigh; ++i)
		{
			const DBL7VECT *c1 = &(centers[s_face_face[startIdx_face + 1 + i]]);
			u[i] = c1->intensity;
			for (int j = 0; j < numNeigh; ++j)
			{
				int startIdx_T = s_face_face[startIdx_face + 1 + j] * numMetricElem;
				const DBL7VECT *c2 = &(centers[s_face_face[startIdx_face + 1 + j]]);
				double p1[2] = { c1->x, c1->y };
				double p2[2] = { c2->x, c2->y };
				double metric[4] = {T[startIdx_T], T[startIdx_T + 1], T[startIdx_T + 2], T[startIdx_T + 3]};
				double r = compute_distance(p1, p2, metric);
				ma[i * numNeigh + j] = phi(r);
			}
		}
		solve(coeff, ma, u, numNeigh); // solve coeff by Gaussian elimination
		// do interpolation
		for (int k1 = 0; k1 < counter; ++k1)
		{
			int m = x[k1];
			int n = y[k1];
			double p1[2] = { (double) m, (double) n };
			double intensity = 0.0;
			for (int k2 = 0; k2 < numNeigh; ++k2)
			{
				const DBL7VECT *center = &(centers[s_face_face[startIdx_face + 1 + k2]]);
				double p2[2] = { center->x, center->y };
				int startIdx_T = s_face_face[startIdx_face + 1 + k2] * numMetricElem;
				double metric[4] = { T[startIdx_T], T[startIdx_T + 1], T[startIdx_T + 2], T[startIdx_T + 3] };
				double r = compute_distance(p1, p2, metric);
				intensity += (coeff[k2] * phi(r));
			}
			F[n*X + m] = intensity; // sum interpolated intensity
		}
		free(ma);
		free(u);
		free(coeff);
	} // end if
}

__global__ void threshold_kernel(double *F, int N)
{
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	if (tx < N)
	{
		if (F[tx] < 0.0)
			F[tx] = 0.0;
		if (F[tx] > 255.0)
			F[tx] = 255.0;
	}
}

__global__ void testKernel(double *x)
{
	double a[] = { 
		1.00, 0.00, 0.00, 0.00, 0.00, 0.00,
		1.00, 0.63, 0.39, 0.25, 0.16, 0.10,
		1.00, 1.26, 1.58, 1.98, 2.49, 3.13,
		1.00, 1.88, 3.55, 6.70, 12.62, 23.80,
		1.00, 2.51, 6.32, 15.88, 39.90, 100.28,
		1.00, 3.14, 9.87, 31.01, 97.41, 306.02 };
	double b[] = { -0.01, 0.61, 0.91, 0.99, 0.60, 0.02 };
	solve(x, a, b, 6);
}

void interpolate_helper(int numMetricElem, int maxNumNeighFace, const double *T, const int *face_face)
{
	size_t bytesImage = sizeof(double) * X * Y; // size of image in bytes
	size_t bytesCenters = sizeof(DBL7VECT) * samples->nf; // size of triangle centers in bytes
	size_t bytesVertices = sizeof(DBL7VECT) * samples->nv; // size of vertices in bytes
	size_t bytesFaces = sizeof(INT3VECT) * samples->nf; // size of faces in bytes
	g_F = (double*) malloc(bytesImage);
	memset(g_F, 0, bytesImage);
	//CUDA_CHECK_RETURN(hipHostAlloc(&g_F, bytesImage, hipHostMallocDefault)); // allocate pinned-memory for g_F on host
	//CUDA_CHECK_RETURN(hipMemset((void*)g_F, 0, bytesImage));
	double *d_F;
	DBL7VECT *d_Centers, *d_Vertices;
	INT3VECT *d_Faces;
	CUDA_CHECK_RETURN(hipMalloc(&d_F, bytesImage)); // allocate memory on device for d_F
	CUDA_CHECK_RETURN(hipMalloc(&d_Centers, bytesCenters)); // allocate memory on device for d_Centers
	CUDA_CHECK_RETURN(hipMalloc(&d_Vertices, bytesVertices)); // allocate memory on device for d_Vertices
	CUDA_CHECK_RETURN(hipMalloc(&d_Faces, bytesFaces)); // allocate memory on device for d_Faces
	CUDA_CHECK_RETURN(hipMemcpy(d_Vertices, samples->vertex, bytesVertices, hipMemcpyHostToDevice)); // copy samples->vertex
	CUDA_CHECK_RETURN(hipMemcpy(d_Faces, samples->face, bytesFaces, hipMemcpyHostToDevice)); // copy samples->face
	CUDA_CHECK_RETURN(hipMemcpy(d_Centers, centers, bytesCenters, hipMemcpyHostToDevice)); // copy centers

	// override default device heap size (8MB)
	CUDA_CHECK_RETURN(hipDeviceSetLimit(hipLimitMallocHeapSize, DEVICE_HEAP));

	// interpolate
	unsigned gridSize = samples->nf / BLOCK_SIZE + ((samples->nf % BLOCK_SIZE) == 0 ? 0 : 1);
	unsigned byteShardMem = sizeof(int) * BLOCK_SIZE * maxNumNeighFace; // size of shared memory in bytes
	interpolate_kernel <<<gridSize, BLOCK_SIZE, byteShardMem>>> (
		d_F, samples->nf, X, Y, numMetricElem, maxNumNeighFace, d_Vertices, d_Faces, d_Centers, T, face_face);
	/*
	double *h_x = (double*)malloc(sizeof(double) * 6);
	double *d_x;
	CUDA_CHECK_RETURN(hipMalloc(&d_x, sizeof(double) * 6));
	testKernel <<<1, 1 >>> (d_x);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipMemcpy(h_x, d_x, sizeof(double) * 6, hipMemcpyDeviceToHost));
	for (int i = 0; i < 6; ++i)
	{
		printf("%g\n", h_x[i]);
	}
	free(h_x);
	CUDA_CHECK_RETURN(hipFree(d_x));*/

	// take average and set threshold
	int newBlockSize = 2 * BLOCK_SIZE;
	gridSize = X * Y / newBlockSize + ((X * Y % newBlockSize) == 0 ? 0 : 1);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	threshold_kernel << <gridSize, newBlockSize >> > (d_F, X * Y);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	// copy result back to host
	CUDA_CHECK_RETURN(hipMemcpy(g_F, d_F, bytesImage, hipMemcpyDeviceToHost));

	

	//for (int j = 0; j < Y; ++j)
	//{
	//	for (int i = 0; i < X; ++i)
	//	{
	//		if ((g_F[j*X + i]) < 0.0)
	//			g_F[j*X + i] = 0.0;
	//		if ((g_F[j*X + i]) > 255.0)
	//			g_F[j*X + i] = 255.0;
	//	}
	//}

	// clean up
	CUDA_CHECK_RETURN(hipFree(d_F));
	CUDA_CHECK_RETURN(hipFree(d_Centers));
	CUDA_CHECK_RETURN(hipFree(d_Vertices));
	CUDA_CHECK_RETURN(hipFree(d_Faces));
}
